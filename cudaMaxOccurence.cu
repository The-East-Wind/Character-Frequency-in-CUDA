#include<stdio.h>
#include<hip/hip_runtime.h>
__global__ void convertToCaps(char *str,int length){
    int index = threadIdx.x+blockIdx.x*blockDim.x;
    if(index<length){
        if(str[index]>=97&&str[index]<=122)
            str[index]-=32;
    }
}
__global__ void findMaxOccurence(char *str,int *count,int length){
    int index = threadIdx.x+blockIdx.x*blockDim.x;
    if(index<length){
        atomicAdd(&count[(int)str[index]-65],1);
    }
}
int countMax(int *count){
    int max=0;
    for(int i=1;i<26;i++){
        if(count[i]>count[max]){
            max=i;
        }
    }
    return max;
}
int main(){
    char *str;
    int n;
    char dummy;
    printf("\nEnter length of string:");
    scanf("%d",&n);
    scanf("%c",&dummy);
    str = (char*)malloc(n*sizeof(char));
    printf("\nEnter the String:");
    scanf("%[^\n]s",str);
    int noOfBlocks = n/1024;
    int noOfThreads;
    noOfBlocks++;
    if(noOfBlocks==1){
        noOfThreads=n;
    }
    else{
        noOfThreads=1024;
    }
    char *dev_str=NULL;int *count;
    hipMallocManaged((void**)&dev_str,n*sizeof(char));
    hipMallocManaged((void**)&count,26*sizeof(int));
    for(int i=0;i<26;i++){
        count[i]=0;
    }
    strcpy(dev_str,str);
    convertToCaps<<<noOfBlocks,noOfThreads>>>(dev_str,n);
    hipDeviceSynchronize();
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    findMaxOccurence<<<noOfBlocks,noOfThreads>>>(dev_str,count,n);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    //printf("\n%s",dev_str);
    int max = countMax(count);
    printf("\nMaximum count = %d",count[max]);
    printf("\nExecution Time = %f ms",milliseconds);
    //printf("%s",str);
    //printf("\n%d",findLen(str));
    return 0;
}